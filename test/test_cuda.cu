#include <stdio.h>
#include <hip/hip_runtime.h>

#include <unistd.h>

#define CUDA_CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("\033[31;31m<%s line:%d %s> ERROR \033[0m",__FUNCTION__,__LINE__,__TIME__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}


__global__ void parallel_add_val(void* dptr,int add,size_t n) {

    char* val = (char*)dptr;
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i==1) {
        // if(val[i]!=0) {
        //     printf("kernel not zero\n");
        // }
        //printf("Yes add-kernel start!\n");
    }
    if(i<n) {
        val[i] += add;
    }
        
}

void test_parallel_add(void* dptr,int add,size_t n) {
    dim3 block(32);
    dim3 grid((n + block.x -1)/block.x);
    //dim3 grid((n)/block.x);
    hipDeviceSynchronize();
    
    parallel_add_val<<<grid,block>>>(dptr,add,n);
    auto func = parallel_add_val;
    printf("func addr:%p\n",(void*)(*func));
    hipDeviceSynchronize();
    //printf("Yes add-kernel finsh!\n");
    void* host_ptr = malloc(n);
    hipMemcpy(host_ptr, dptr, n, hipMemcpyDeviceToHost);
    //check
    for(int i=0;i<n;i+=10) {
        if( ((char*)host_ptr)[i] != add ) {
            printf("check fail,%d",i);
            exit(-1);
        }
    }
    printf("%p %p %p\n",&dptr,nullptr,&n);
    printf("%ld %ld\n",sizeof(dptr),sizeof(n));
    parallel_add_val<<<grid,block>>>(dptr,-1*add,n);
    printf("check pass!\n");
}

int main() {

    // auto ret = cuInit(0);
    // printf("init ret:%d\n",ret);
    // CUdeviceptr cudptr;
    // cuMemAlloc(&cudptr,1024);
    void* dptr;
    size_t size = 1024;
    auto res = hipMalloc(&dptr,size);
    printf("%d\n",res);
    // getchar();
    // for(int i=0;i<1;i++) {
    //     test_parallel_add(dptr,1, size);
    //     usleep(1000);
    // }
    
    return 0;
}
