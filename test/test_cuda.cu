#include <stdio.h>
#include <hip/hip_runtime.h>

#include <unistd.h>

#define CUDA_CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("\033[31;31m<%s line:%d %s> ERROR \033[0m",__FUNCTION__,__LINE__,__TIME__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}


__global__ void parallel_add_val(void* dptr,int add,size_t n) {

    char* val = (char*)dptr;
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i==1) {
        // if(val[i]!=0) {
        //     printf("kernel not zero\n");
        // }
        //printf("Yes add-kernel start!\n");
    }
    if(i<n) {
        val[i] += add;
    }
        
}

void test_parallel_add(void* dptr,int add,size_t n) {
    dim3 block(32);
    dim3 grid((n + block.x -1)/block.x);
    parallel_add_val<<<grid,block>>>(dptr,add,n);
    hipDeviceSynchronize();
    //printf("Yes add-kernel finsh!\n");
    void* host_ptr = malloc(n);
    hipMemcpy(host_ptr, dptr, n, hipMemcpyDeviceToHost);
    //check
    for(int i=0;i<n;i+=10) {
        if( ((char*)host_ptr)[i] != add ) {
            printf("check fail,%d",i);
            exit(-1);
        }
    }
    parallel_add_val<<<grid,block>>>(dptr,-1*add,n);
    printf("check pass!\n");
}

int main() {

    void* dptr;
    size_t size = 1<<20;
    hipMalloc(&dptr,size);
    //getchar();
    //for(int i=0;i<10;i++) {
    test_parallel_add(dptr,1, size);
    usleep(1000);
    //}
    
    return 0;
}