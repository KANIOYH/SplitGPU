#include <stdio.h>
#include <hip/hip_runtime.h>

#include <unistd.h>

#define CUDA_CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("\033[31;31m<%s line:%d %s> ERROR \033[0m",__FUNCTION__,__LINE__,__TIME__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}


__global__ void parallel_add_val(void* dptr,int add,size_t n) {

    char* val = (char*)dptr;
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i==1) {
        // if(val[i]!=0) {
        //     printf("kernel not zero\n");
        // }
        //printf("Yes add-kernel start!\n");
    }
    if(i<n) {
        val[i] += add;
    }
        
}

void test_parallel_add(void* dptr,int add,size_t n) {
    dim3 block(32);
    dim3 grid((n + block.x -1)/block.x);
    //dim3 grid((n)/block.x);
    hipDeviceSynchronize();
    
    parallel_add_val<<<grid,block>>>(dptr,add,n);
    auto func = parallel_add_val;
    printf("func addr:%p\n",(void*)(*func));
    hipDeviceSynchronize();
    //printf("Yes add-kernel finsh!\n");
    void* host_ptr = malloc(n);
    hipMemcpy(host_ptr, dptr, n, hipMemcpyDeviceToHost);
    //check
    for(int i=0;i<n;i+=10) {
        if( ((char*)host_ptr)[i] != add ) {
            printf("check fail,%d",i);
            exit(-1);
        }
    }
    printf("%p %p %p\n",&dptr,nullptr,&n);
    printf("%ld %ld\n",sizeof(dptr),sizeof(n));
    parallel_add_val<<<grid,block>>>(dptr,-1*add,n);
    printf("check pass!\n");
}

int main() {

    // auto ret = cuInit(0);
    // printf("init ret:%d\n",ret);
    // CUdeviceptr cudptr;
    // cuMemAlloc(&cudptr,1024);
    void* dptr, *dptr2;
    char* ptr,*cptr;
    size_t size = 1024;
    auto res = hipMalloc(&dptr,size);
    res = hipMalloc(&dptr2,size);
    ptr = (char*)malloc(size);
    cptr = (char*)malloc(size);
    for(int i=0;i<size;i++) {
        ptr[i] = i%10;
    }
    printf("res %d\n",res);
    res = hipMemcpy(dptr, ptr, size, hipMemcpyHostToDevice);
    printf("res %d\n",res);
    res = hipMemcpy(cptr, dptr, size, hipMemcpyDeviceToHost);
    printf("res %d\n",res);
    for(int i=0;i<size;i++) {
        if(cptr[i] != i%10) {
            printf("fail,i %d, cptr:%d\n",i,cptr[i]);
            exit(0);
        }
    }
    printf("pass!\n");
    return 0;
}
