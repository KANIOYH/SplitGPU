#include <stdio.h>
#include <hip/hip_runtime.h>

#include <unistd.h>

#define CUDA_CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("\033[31;31m<%s line:%d %s> ERROR \033[0m",__FUNCTION__,__LINE__,__TIME__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}


__global__ void parallel_add_val(void* dptr,int add,size_t n) {
    char* val = (char*)dptr;
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n) {
        val[i] += add;
    }
}

void test_parallel_add(void* dptr,int add,size_t n) {
    dim3 block(32);
    dim3 grid((n + block.x -1)/block.x);
    parallel_add_val<<<grid,block>>>(dptr,add,n);
    hipDeviceSynchronize();
    void* host_ptr = malloc(n);
    hipMemcpy(host_ptr, dptr, n, hipMemcpyDeviceToHost);
    //check
    for(int i=0;i<n;i+=10) {
        if( ((char*)host_ptr)[i] != add ) {
            printf("check fail,%d",i);
            exit(-1);
        }
    }
    parallel_add_val<<<grid,block>>>(dptr,-1*add,n);
    printf("check pass!\n");
}

int main() {
    void* dptr;
    size_t size = 1<<20;
    hipMalloc(&dptr,size);
    test_parallel_add(dptr,1, size);
    return 0;
}