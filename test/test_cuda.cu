#include <stdio.h>
#include <hip/hip_runtime.h>

#include <unistd.h>

#define CUDA_CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("\033[31;31m<%s line:%d %s> ERROR \033[0m",__FUNCTION__,__LINE__,__TIME__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

struct haha {
    double a;
    bool b;
    // int b;
};

__global__ void parallel_hello(haha ja,int a,void* dptr) {

    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i==1)
        printf("hello val\n");
    
        
}

__global__ void parallel_add_val(void* dptr,int add ,size_t n) {

    char* val = (char*)dptr;
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i==1) {
        // if(val[i]!=0) {
        //     printf("kernel not zero\n");
        // }
        printf("Yes add-kernel start!\n");
    }
    if(i<n) {
        val[i] += add;
    }
        
}

void test_parallel_add(void* dptr,int add,size_t n) {
    dim3 block(32);
    dim3 grid((n + block.x -1)/block.x);
    //parallel_hello<<<grid,block>>>(1);
    parallel_add_val<<<grid,block>>>(dptr, add ,n);
    // printf("dptr\n");
    // for(int i=0;i<sizeof(dptr);i++) {
    //     printf("%02x ",((unsigned char*)(&dptr))[i]);
    // }
    // printf("\nadd\n");
    // for(int i=0;i<sizeof(dptr);i++) {
    //     printf("%02x ",((unsigned char*)(&add))[i]);
    // }
    // printf("\nn\n");
    // for(int i=0;i<sizeof(dptr);i++) {
    //     printf("%02x ",((unsigned char*)(&n))[i]);
    // }
    hipDeviceSynchronize();
    // //printf("Yes add-kernel finsh!\n");
    // void* host_ptr = malloc(n);
    // cudaMemcpy(host_ptr, dptr, n, cudaMemcpyDeviceToHost);
    // //check
    // for(int i=0;i<n;i+=10) {
    //     if( ((char*)host_ptr)[i] != add ) {
    //         printf("check fail,%d",i);
    //         exit(-1);
    //     }
    // }
    // printf("%p %p %p\n",&dptr,nullptr,&n);
    // printf("%ld %ld\n",sizeof(dptr),sizeof(n));
    // parallel_add_val<<<grid,block>>>(dptr,-1*add,n);
    // printf("check pass!\n");
}

int main() {

    // auto ret = cuInit(0);
    // printf("init ret:%d\n",ret);
    // CUdeviceptr cudptr;
    // cuMemAlloc(&cudptr,1024);
    void* dptr;
    char* ptr,*cptr;
    size_t size = 1024;
    auto res = hipMalloc(&dptr,size);
    ptr = (char*)malloc(size);
    cptr = (char*)malloc(size);
    for(int i=0;i<size;i++) {
        ptr[i] = i%10;
    }
    printf("res %d\n",res);
    res = hipMemcpy(dptr, ptr, size, hipMemcpyHostToDevice);
    printf("res %d\n",res);
    res = hipMemcpy(cptr, dptr, size, hipMemcpyDeviceToHost);
    printf("res %d\n",res);
    for(int i=0;i<size;i++) {
        if(cptr[i] != i%10) {
            printf("fail,i %d, cptr:%d\n",i,cptr[i]);
            exit(0);
        }
    }
    printf("pass!\n");
    int add = 1;
    test_parallel_add(dptr,add,size);
    return 0;
}
