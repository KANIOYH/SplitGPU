#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <vector>
int main() {
    int N = 128;
    size_t size = N * sizeof(float);
    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);
    float *h_Cb = (float *)malloc(size);

    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = i;
        h_C[i] = h_A[i]+h_B[i]+1;
    }
    hipError_t res;
    hipInit(0);
    hipDevice_t device;
    res = hipDeviceGet(&device, 0);
    if (res != hipSuccess){
        printf("hipDeviceGet\n");
        exit(EXIT_FAILURE);
    }
    hipCtx_t context;
    res = hipCtxCreate(&context, 0, device);
    if (res != hipSuccess){
        printf("hipCtxCreate\n");
        exit(EXIT_FAILURE);
    }

    hipDeviceptr_t d_A;
    hipMalloc(&d_A, size);
    hipDeviceptr_t d_B;
    hipMalloc(&d_B, size);
    hipDeviceptr_t d_C;
    hipMalloc(&d_C, size);
    hipMemcpyHtoD(d_A, h_A, size);
    hipMemcpyHtoD(d_B, h_B, size);
    hipModule_t module;
    res = hipModuleLoad(&module, "/home/chenyuanhui/project/SplitGPU/data.fatbin");
    if (res != hipSuccess){
        printf("hipModuleLoad\n");
        exit(EXIT_FAILURE);
    }    
    hipFunction_t f;
    res = hipModuleGetFunction(&f, module, "_Z14parallel_helloi");
    if (res != hipSuccess){
        printf("hipModuleGetFunction\n");
        exit(EXIT_FAILURE);
    }    
    unsigned char* header = (unsigned char*)f;
    int threadsPerBlock = 256;
    int blocksPerGrid =
            (N + threadsPerBlock - 1) / threadsPerBlock;
    int val = 1;
    void* args[] = { &val};
    hipModuleLaunchKernel(f,
                   blocksPerGrid, 1, 1, threadsPerBlock, 1, 1,
                   0, 0, args, 0);
    hipCtxSynchronize();

    // hipModule_t module;
    // res = hipModuleLoad(&module, "/home/chenyuanhui/project/cuda_hook/sample/cuda/vector_add.fatbin");
    // if (res != hipSuccess){
    //     printf("hipModuleLoad\n");
    //     exit(EXIT_FAILURE);
    // }    
    // hipFunction_t vecAdd;
    // res = hipModuleGetFunction(&vecAdd, module, "_Z9vectorAddPKfS0_Pfi");
    // if (res != hipSuccess){
    //     printf("hipModuleGetFunction\n");
    //     exit(EXIT_FAILURE);
    // }    
    // unsigned char* header = (unsigned char*)vecAdd;

    // for(int i=0;i<32;i++) {
    //     if(i%4==0)
    //         printf("\n");
    //     unsigned char v = header[i];
    //     printf("%02x ",v);
    // }
    // int threadsPerBlock = 256;
    // int blocksPerGrid =
    //         (N + threadsPerBlock - 1) / threadsPerBlock;

    // void* args[] = { &d_A, &d_B, &d_C, &N ,&threadsPerBlock};
    // hipModuleLaunchKernel(vecAdd,
    //                blocksPerGrid, 1, 1, threadsPerBlock, 1, 1,
    //                0, 0, args, 0);
    // hipCtxSynchronize();
    // hipMemcpyDtoH(h_Cb, d_C, size);
    // for (int i = 0; i < N; ++i) {
    //     if( abs(h_Cb[i]-h_C[i]) > 0.01 ) {
    //         printf("umatch %f %f %d\n",h_Cb[i],h_C[i],i);
    //         exit(-1);
    //     }
    // }
    printf("aaa check pass!\n");
    return 0;
}