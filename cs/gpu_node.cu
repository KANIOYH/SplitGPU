#include <cstddef>
#include <cstring>
#include <string>
#include <sys/types.h>
#include "gpu_node.h"
#include "cuda_param.h"

namespace SplitGPU {



GPU_node::GPU_node() {

    for(int i=SYMBOL_START;i<SYMBOL_END;i++) {
        func_id_map[cuda_func_name[i]] = i;
    }

    /* both using cuda driver api and cuda runtime api */
    hipInit(0);
    hipDeviceGet(&device, 0);
    hipSetDevice(0);
    hipCtxGetCurrent(&context);

}

void GPU_node::exec_cuda(std::string cuda_name,char args[]) {

    auto item = func_id_map.find(cuda_name);
    if(item == func_id_map.end()) {
        printf("not find\n");
        return;
    }
        
    off_t offset = 0;
    switch (item->second) {
    case SYMBOL_cudaMalloc: {
        void** p_devPtr;
        size_t* p_size;
        UNPACK_PARAM(args,offset,p_devPtr);
        UNPACK_PARAM(args,offset,p_size);
        hipError_t res = hipMalloc(p_devPtr, *p_size);
        memcpy(&args[offset], &res, sizeof(res));
        printf("hipMalloc\n");
    } break;
    case SYMBOL_cudaFree: {

    } break;
    case SYMBOL_cudaDeviceSynchronize: {
        hipError_t res = hipDeviceSynchronize();
        memcpy(&args[offset], &res, sizeof(res));
    };
    case SYMBOL_cudaMemcpy: {

    } break;
    default: {
        /*unregister cuda function*/
    }

    }
}

}