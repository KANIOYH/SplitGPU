#include <cstddef>
#include <cstring>
#include <string>
#include <sys/types.h>
#include "gpu_node.h"
#include "cuda_param.h"

namespace SplitGPU {



GPU_node::GPU_node() {
    /* both using cuda driver api and cuda runtime api */
    hipInit(0);
    hipDeviceGet(&device, 0);
    hipSetDevice(0);
    hipCtxGetCurrent(&context);
}

// void GPU_node::exec_cuda(std::string cuda_name,gpu_request* req, hipError_t& result) {
//     char* args = req->fargs;
//     printf("%s\n",cuda_name.c_str());
//     auto item = func_id_map.find(cuda_name);
//     if(item == func_id_map.end()) {
//         printf("not find\n");
//         return;
//     }
        
//     off_t offset = 0;
//     switch (item->second) {
//     case SYMBOL_cudaMalloc: {
//         void** p_devPtr;
//         size_t* p_size;
//         UNPACK_PARAM(args,offset,p_devPtr);
//         UNPACK_PARAM(args,offset,p_size);
//         result = hipMalloc(p_devPtr, *p_size);
//         memcpy(&args[offset], &result, sizeof(result));
//         printf("hipMalloc,%p %ld\n",*p_devPtr,*p_size);
//     } break;
//     case SYMBOL_cudaFree: {

//     } break;
//     case SYMBOL_cudaDeviceSynchronize: {
//         result = hipDeviceSynchronize();
//         memcpy(&args[offset], &result, sizeof(result));
//     };
//     case SYMBOL_cudaMemcpy: {
//         char** p_dst;
//         char** p_src;
//         size_t* p_count;
//         hipMemcpyKind* p_kind;
//         UNPACK_PARAM(args,offset,p_dst)
//         UNPACK_PARAM(args,offset,p_src)
//         UNPACK_PARAM(args,offset,p_count)
//         UNPACK_PARAM(args,offset,p_kind)
//         switch(*p_kind) {
//         case hipMemcpyHostToHost:
//         case hipMemcpyHostToDevice: {
//             printf("host to device\n");
//             result = hipMemcpy(*p_dst, req->extra_data,
//                                              *p_count, *p_kind);
//             memcpy(&args[offset], &result, sizeof(result));
//         } break;
//         case hipMemcpyDeviceToHost:
//         case hipMemcpyDeviceToDevice:
//         case hipMemcpyDefault:
//           break;
//         }
//     } break;
//     default: {
//         /*unregister cuda function*/
//     }

//     }
// }

hipError_t exec_cudaMalloc(void** devPtr, size_t size) {
    hipError_t result;
    result = hipMalloc(devPtr,size);
    return result;
}
hipError_t exec_cudaDeviceSynchronize() {
    hipError_t result = hipSuccess;;
    return result;
}
hipError_t exec_cudaMemcpy(void* dst,void* src, size_t size, hipMemcpyKind kind) {
    hipError_t result = hipSuccess;
    return result;   
}

}