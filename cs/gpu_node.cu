#include <cstddef>
#include <cstring>
#include <string>
#include <sys/types.h>
#include "gpu_node.h"
#include "cuda_param.h"

namespace SplitGPU {



GPU_node::GPU_node() {
    /* both using cuda driver api and cuda runtime api */
//     hipError_t res;
//     hipInit(0);
//     hipDeviceGet(&device, 0);
//     hipSetDevice(0);
//     res = hipCtxGetCurrent(&context);
//     if (res != hipSuccess){
//         printf("hipCtxGetCurrent\n");
//         exit(EXIT_FAILURE);
//     }  
//     res = hipModuleLoad(&module, "/home/chenyuanhui/project/SplitGPU/data.fatbin");
//     if (res != hipSuccess){
//         printf("hipModuleLoad\n");
//         exit(EXIT_FAILURE);
//     }   
}

hipFunction_t GPU_node::get_func(std::string func_name) {
    hipFunction_t func;
    printf("func name:%s\n",func_name.c_str());
    auto res = hipModuleGetFunction(&func, module, "_Z16parallel_add_valPvim");
    if (res != hipSuccess){
        printf("hipModuleGetFunction\n");
        exit(EXIT_FAILURE);
    }  
    return func;
}

hipError_t GPU_node::exec_cudaMalloc(void** devPtr, size_t size) {
    hipError_t result;
    result = hipMalloc(devPtr,size);
    return result;
}
hipError_t GPU_node::exec_cudaDeviceSynchronize() {
    hipError_t result = hipSuccess;;
    return result;
}
hipError_t GPU_node::exec_cudaMemcpy(void* dst,void* src, size_t size, hipMemcpyKind kind) {
    hipError_t result = hipSuccess;
    return result;   
}

hipError_t GPU_node::exec_kernel(std::string func_name,dim3 gridDim, dim3 blockDim, void **args,size_t sharedMem, hipStream_t stream) {
    hipError_t res;
    hipInit(0);
    hipDeviceGet(&device, 0);
    hipSetDevice(0);
    res = hipCtxGetCurrent(&context);
    if (res != hipSuccess){
        printf("hipCtxGetCurrent\n");
        exit(EXIT_FAILURE);
    }  

    hipError_t result = hipSuccess;
    hipError_t curesult;
    //hipDeviceptr_t dptr = (hipDeviceptr_t)args;
    printf("------------\n");
    for(int i=0;i<156;i++) {
        printf("%02x ",((unsigned char*)args)[i]);
        if(i!=0 && i%8==0)
            printf("\n");
    }
    printf("\n------------\n");
    hipFunction_t f;
    res = hipModuleGetFunction(&f, module, func_name.data());
    if (res != hipSuccess){
        printf("hipModuleGetFunction\n");
        exit(EXIT_FAILURE);
    }    
    curesult = hipModuleLaunchKernel(f,gridDim.x, gridDim.y, gridDim.z,
                         blockDim.x, blockDim.y, blockDim.z,
                   sharedMem, stream, 0,(void**)&args);
    hipCtxSynchronize();
    // int threadsPerBlock = 256;
    // int blocksPerGrid =(128 + threadsPerBlock - 1) / threadsPerBlock;
    // int val = 1;
    // void* arg[] = {&val};
    // curesult = hipModuleLaunchKernel(kernel,
    //                blocksPerGrid, 1, 1, threadsPerBlock, 1, 1,
    //                0, 0, arg, 0);
    if(curesult!=hipSuccess) {
        printf("launch err:%d\n",curesult);
    // hipCtxSynchronize();
    } else {
        printf("launch ok\n");
    }
    return result;   
}

}